#include <hip/hip_runtime.h>
#include <dune/common/memory/cuda_allocator.hh>

using namespace Dune;
using namespace Dune::Memory;
using namespace Dune::Cuda;

template <typename DT_>
typename std::allocator<DT_>::pointer CudaAllocator<DT_>::allocate(size_t n, typename std::allocator<void>::const_pointer /*hint*/)
{
  void * r;
  hipError_t status = hipMalloc(&r, n * sizeof(DT_));
  if (status != hipSuccess)
    throw new std::bad_alloc;

  return (DT_*)r;
}

template <typename DT_>
void CudaAllocator<DT_>::deallocate(typename std::allocator<DT_>::pointer p, size_t /*n*/)
{
  hipFree((void*) p);
}

template typename std::allocator<float>::pointer CudaAllocator<float>::allocate(size_t n, typename std::allocator<void>::const_pointer);
template typename std::allocator<double>::pointer CudaAllocator<double>::allocate(size_t n, typename std::allocator<void>::const_pointer);
template void CudaAllocator<float>::deallocate(typename std::allocator<float>::pointer, size_t);
template void CudaAllocator<double>::deallocate(typename std::allocator<double>::pointer, size_t);

template <typename DT_>
void Dune::Cuda::upload(DT_ * dst, const DT_ * src, size_t count)
{
  hipMemcpy(dst, src, count * sizeof(DT_), hipMemcpyHostToDevice);
}

template <typename DT_>
void Dune::Cuda::download(DT_ * dst, const DT_ * src, size_t count)
{
  hipMemcpy(dst, src, count * sizeof(DT_), hipMemcpyDeviceToHost);
}

template <typename DT_>
void Dune::Cuda::copy(DT_ * dst, const DT_ * src, size_t count)
{
  hipMemcpy(dst, src, count * sizeof(DT_), hipMemcpyDeviceToDevice);
}

template <typename DT_>
void Dune::Cuda::set(DT_ * dst, const DT_ & val)
{
  hipMemcpy(dst, &val, sizeof(DT_), hipMemcpyHostToDevice);
}

template <typename DT_>
DT_ Dune::Cuda::get(DT_ * src)
{
  DT_ result;
  hipMemcpy(&result, src, sizeof(DT_), hipMemcpyDeviceToHost);
  return result;
}

template void Dune::Cuda::upload(float *, const float *, size_t);
template void Dune::Cuda::upload(double *, const double *, size_t);
template void Dune::Cuda::download(float *, const float *, size_t);
template void Dune::Cuda::download(double *, const double *, size_t);
template void Dune::Cuda::copy(float *, const float *, size_t);
template void Dune::Cuda::copy(double *, const double *, size_t);
template void Dune::Cuda::set(float *, const float &);
template void Dune::Cuda::set(double *, const double &);
template float Dune::Cuda::get(float *);
template double Dune::Cuda::get(double *);
