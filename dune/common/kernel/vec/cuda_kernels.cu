// -*- tab-width: 4; indent-tabs-mode: nil; c-basic-offset: 2 -*-
// vi: set et ts=4 sw=2 sts=2:


#include <hip/hip_runtime.h>
#include <hipblas.h>

namespace Dune
{
  namespace Cuda
  {
    //-------------- sum ---------------
    template <typename DT_>
    __global__ void device_sum(DT_ * r, const DT_ * x, const DT_ * y, const unsigned long count)
    {
      unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      if (idx >= count)
        return;
      r[idx] = x[idx] + y[idx];
    }

    template <typename DT_>
    DT_ * sum(DT_ * r, const DT_* x, const DT_ * y, const unsigned long size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((size)/(double)(block.x));
      device_sum<<<grid, block>>>(r, x, y, size);
      return r;
    }

    template float* sum(float*, const float*, const float*, unsigned long);
    template double* sum(double*, const double*, const double*, unsigned long);

    //-------------- difference ---------------
    template <typename DT_>
    __global__ void device_difference(DT_ * r, const DT_ * x, const DT_ * y, const unsigned long count)
    {
      unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      if (idx >= count)
        return;
      r[idx] = x[idx] - y[idx];
    }

    template <typename DT_>
    DT_ * difference(DT_ * r, const DT_* x, const DT_ * y, const unsigned long size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((size)/(double)(block.x));
      device_difference<<<grid, block>>>(r, x, y, size);
      return r;
    }

    template float* difference(float*, const float*, const float*, unsigned long);
    template double* difference(double*, const double*, const double*, unsigned long);

    //-------------- element_product ---------------
    template <typename DT_>
    __global__ void device_element_product(DT_ * r, const DT_ * x, const DT_ * y, const unsigned long count)
    {
      unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      if (idx >= count)
        return;
      r[idx] = x[idx] * y[idx];
    }

    template <typename DT_>
    DT_ * element_product(DT_ * r, const DT_* x, const DT_ * y, const unsigned long size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((size)/(double)(block.x));
      device_element_product<<<grid, block>>>(r, x, y, size);
      return r;
    }

    template float* element_product(float*, const float*, const float*, unsigned long);
    template double* element_product(double*, const double*, const double*, unsigned long);

    //-------------- element_division ---------------
    template <typename DT_>
    __global__ void device_element_division(DT_ * r, const DT_ * x, const DT_ * y, const unsigned long count)
    {
      unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      if (idx >= count)
        return;
      r[idx] = x[idx] / y[idx];
    }

    template <typename DT_>
    DT_ * element_division(DT_ * r, const DT_* x, const DT_ * y, const unsigned long size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((size)/(double)(block.x));
      device_element_division<<<grid, block>>>(r, x, y, size);
      return r;
    }

    template float* element_division(float*, const float*, const float*, unsigned long);
    template double* element_division(double*, const double*, const double*, unsigned long);

    //-------------- sum_scalar ---------------
    template <typename DT_>
    __global__ void device_sum_scalar(DT_ * r, const DT_ * x, DT_ y, const unsigned long count)
    {
      unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      if (idx >= count)
        return;
      r[idx] = x[idx] + y;
    }

    template <typename DT_>
    DT_ * sum_scalar(DT_ * r, const DT_* x, DT_ y, const unsigned long size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((size)/(double)(block.x));
      device_sum_scalar<<<grid, block>>>(r, x, y, size);
      return r;
    }

    template float* sum_scalar(float*, const float*, float, unsigned long);
    template double* sum_scalar(double*, const double*, double, unsigned long);

    //-------------- difference_scalar ---------------
    template <typename DT_>
    __global__ void device_difference_scalar(DT_ * r, const DT_ * x, DT_ y, const unsigned long count)
    {
      unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      if (idx >= count)
        return;
      r[idx] = x[idx] - y;
    }

    template <typename DT_>
    DT_ * difference_scalar(DT_ * r, const DT_* x, DT_ y, const unsigned long size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((size)/(double)(block.x));
      device_difference_scalar<<<grid, block>>>(r, x, y, size);
      return r;
    }

    template float* difference_scalar(float*, const float*, float, unsigned long);
    template double* difference_scalar(double*, const double*, double, unsigned long);

    //-------------- product_scalar ---------------
    template <typename DT_>
    __global__ void device_product_scalar(DT_ * r, const DT_ * x, DT_ y, const unsigned long count)
    {
      unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      if (idx >= count)
        return;
      r[idx] = x[idx] * y;
    }

    template <typename DT_>
    DT_ * product_scalar(DT_ * r, const DT_* x, DT_ y, const unsigned long size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((size)/(double)(block.x));
      device_product_scalar<<<grid, block>>>(r, x, y, size);
      return r;
    }

    template float* product_scalar(float*, const float*, float, unsigned long);
    template double* product_scalar(double*, const double*, double, unsigned long);

    //-------------- division_scalar ---------------
    template <typename DT_>
    __global__ void device_division_scalar(DT_ * r, const DT_ * x, DT_ y, const unsigned long count)
    {
      unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      if (idx >= count)
        return;
      r[idx] = x[idx] / y;
    }

    template <typename DT_>
    DT_ * division_scalar(DT_ * r, const DT_* x, DT_ y, const unsigned long size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((size)/(double)(block.x));
      device_division_scalar<<<grid, block>>>(r, x, y, size);
      return r;
    }

    template float* division_scalar(float*, const float*, float, unsigned long);
    template double* division_scalar(double*, const double*, double, unsigned long);

    //-------------- axpy ---------------
    template <typename DT_>
    __global__ void device_axpy(DT_ * r, const DT_ * x, DT_ a, const DT_ * y, const unsigned long count)
    {
      unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      if (idx >= count)
        return;
      r[idx] = x[idx] * a + y[idx];
    }

    template <typename DT_>
    DT_ * axpy(DT_ * r, const DT_* x, DT_ a, const DT_ * y, const unsigned long size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((size)/(double)(block.x));
      device_axpy<<<grid, block>>>(r, x, a, y, size);
      return r;
    }

    template float* axpy(float*, const float*, float, const float*, unsigned long);
    template double* axpy(double*, const double*, double, const double*, unsigned long);

    //-------------- dot ---------------
    float device_dot(const float * x, const float * y, const unsigned long size)
    {
        return cublasSdot(size, x, 1, y, 1);
    }

    double device_dot(const double * x, const double * y, const unsigned long size)
    {
        return cublasDdot(size, x, 1, y, 1);
    }

    template <typename DT_>
    DT_ dot(const DT_ * x, const DT_* y, const unsigned long size)
    {
      cublasInit();
      DT_ r = device_dot(x, y, size);
      cublasShutdown();
      return r;
    }

    template float dot(const float*, const float*, unsigned long);
    template double dot(const double*, const double*, unsigned long);

    //-------------- two_norm2 ---------------
    float device_two_norm2(const float * x, const unsigned long size)
    {
        return cublasSnrm2(size, x, 1);
    }

    double device_two_norm2(const double * x, const unsigned long size)
    {
        return cublasDnrm2(size, x, 1);
    }

    template <typename DT_>
    DT_ two_norm2(const DT_ * x, const unsigned long size)
    {
      cublasInit();
      DT_ r = device_two_norm2(x, size);
      cublasShutdown();
      return r;
    }

    template float two_norm2(const float*, unsigned long);
    template double two_norm2(const double*, unsigned long);

    //-------------- one_norm ---------------
    float device_one_norm(const float * x, const unsigned long size)
    {
        return cublasSasum(size, x, 1);
    }

    double device_one_norm(const double * x, const unsigned long size)
    {
        return cublasDasum(size, x, 1);
    }

    template <typename DT_>
    DT_ one_norm(const DT_ * x, const unsigned long size)
    {
      cublasInit();
      DT_ r = device_one_norm(x, size);
      cublasShutdown();
      return r;
    }

    template float one_norm(const float*, unsigned long);
    template double one_norm(const double*, unsigned long);

    //-------------- infinity_norm ---------------
    int device_infinity_norm(const float * x, const unsigned long size)
    {
        return cublasIsamax(size, x, 1) - 1;
    }

    int device_infinity_norm(const double * x, const unsigned long size)
    {
        return cublasIdamax(size, x, 1) - 1;
    }

    template <typename DT_>
    DT_ infinity_norm(const DT_ * x, const unsigned long size)
    {
      cublasInit();
      int i = device_infinity_norm(x, size);
      cublasShutdown();
      DT_ r;
      hipMemcpy(&r, x + i, sizeof(DT_), hipMemcpyDeviceToHost);
      return r;
    }

    template float infinity_norm(const float*, unsigned long);
    template double infinity_norm(const double*, unsigned long);
  }
}
