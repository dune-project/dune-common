// -*- tab-width: 4; indent-tabs-mode: nil; c-basic-offset: 2 -*-
// vi: set et ts=4 sw=2 sts=2:


#include <hip/hip_runtime.h>
#include <hipblas.h>

namespace Dune
{
  namespace Cuda
  {
    //-------------- mv ---------------
    template <typename DT_>
    __global__ void device_mv(const DT_ * x, DT_ * y, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      const unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      const unsigned long row = idx;
      if (row >= rows)
        return;

      DT_ r(0);
      const unsigned long chunk(row / rows_per_chunk);
      const unsigned long local_row(row % rows_per_chunk);
      const unsigned long chunk_end( (chunk == chunks - 1) ? allocated_size : cs[chunk+1]);

      for (unsigned long pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol += rows_per_chunk)
          r += x[col[pcol]] * data[pcol];
      y[row] = r;
    }

    template <typename DT_>
    void mv(const DT_ * x, DT_ * y, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((rows)/(double)(block.x));
      device_mv<<<grid, block>>>(x, y, data, cs, col, rows, rows_per_chunk, chunks, allocated_size);
    }

    template void mv(const float*, float*, const float*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);
    template void mv(const double*, double*, const double*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);

    //-------------- umv ---------------
    template <typename DT_>
    __global__ void device_umv(const DT_ * x, DT_ * y, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      const unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      const unsigned long row = idx;
      if (row >= rows)
        return;

      DT_ r(y[row]);
      const unsigned long chunk(row / rows_per_chunk);
      const unsigned long local_row(row % rows_per_chunk);
      const unsigned long chunk_end( (chunk == chunks - 1) ? allocated_size : cs[chunk+1]);

      for (unsigned long pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol += rows_per_chunk)
          r += x[col[pcol]] * data[pcol];
      y[row] = r;
    }

    template <typename DT_>
    void umv(const DT_ * x, DT_ * y, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((rows)/(double)(block.x));
      device_umv<<<grid, block>>>(x, y, data, cs, col, rows, rows_per_chunk, chunks, allocated_size);
    }

    template void umv(const float*, float*, const float*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);
    template void umv(const double*, double*, const double*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);

    //-------------- mmv ---------------
    template <typename DT_>
    __global__ void device_mmv(const DT_ * x, DT_ * y, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      const unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      const unsigned long row = idx;
      if (row >= rows)
        return;

      DT_ r(y[row]);
      const unsigned long chunk(row / rows_per_chunk);
      const unsigned long local_row(row % rows_per_chunk);
      const unsigned long chunk_end( (chunk == chunks - 1) ? allocated_size : cs[chunk+1]);

      for (unsigned long pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol += rows_per_chunk)
          r -= x[col[pcol]] * data[pcol];
      y[row] = r;
    }

    template <typename DT_>
    void mmv(const DT_ * x, DT_ * y, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((rows)/(double)(block.x));
      device_mmv<<<grid, block>>>(x, y, data, cs, col, rows, rows_per_chunk, chunks, allocated_size);
    }

    template void mmv(const float*, float*, const float*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);
    template void mmv(const double*, double*, const double*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);

    //-------------- usmv ---------------
    template <typename DT_>
    __global__ void device_usmv(DT_ alpha, const DT_ * x, DT_ * y, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      const unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      const unsigned long row = idx;
      if (row >= rows)
        return;

      DT_ r(y[row]);
      const unsigned long chunk(row / rows_per_chunk);
      const unsigned long local_row(row % rows_per_chunk);
      const unsigned long chunk_end( (chunk == chunks - 1) ? allocated_size : cs[chunk+1]);

      for (unsigned long pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol += rows_per_chunk)
          r += x[col[pcol]] * data[pcol];
      y[row] = r * alpha;
    }

    template <typename DT_>
    void usmv(DT_ alpha, const DT_ * x, DT_ * y, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((rows)/(double)(block.x));
      device_usmv<<<grid, block>>>(alpha, x, y, data, cs, col, rows, rows_per_chunk, chunks, allocated_size);
    }

    template void usmv(float, const float*, float*, const float*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);
    template void usmv(double, const double*, double*, const double*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);

    //-------------- sequential_jacobi ---------------
    template <typename DT_>
    __global__ void device_sequential_jacobi(const DT_ * v, const DT_ * d, DT_ * v_new, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      const unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      const unsigned long row = idx;
      if (row >= rows)
        return;

      DT_ rhs(d[row]);
      DT_ diag(DT_(1));
      const unsigned long chunk(row / rows_per_chunk);
      const unsigned long local_row(row % rows_per_chunk);
      const unsigned long chunk_end( (chunk == chunks - 1) ? allocated_size : cs[chunk+1]);

      for (unsigned long pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol += rows_per_chunk)
      {
        if (row == col[pcol])
        {
          diag = data[pcol];
        }
        else
        {
          rhs -= v[col[pcol]] * data[pcol];
        }
      }
      diag = (diag == DT_(0)) ? DT_(1) : diag;
      v_new[row] = rhs / diag;
    }

    template <typename DT_>
    void sequential_jacobi(const DT_ * v, const DT_ * d, DT_ * v_new,
        const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((rows)/(double)(block.x));
      device_sequential_jacobi<<<grid, block>>>(v, d, v_new, data, cs, col, rows, rows_per_chunk, chunks, allocated_size);
    }

    template void sequential_jacobi(const float*, const float*, float*, const float*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);
    template void sequential_jacobi(const double*, const double*, double*, const double*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);
  }
}
