// -*- tab-width: 4; indent-tabs-mode: nil; c-basic-offset: 2 -*-
// vi: set et ts=4 sw=2 sts=2:


#include <hip/hip_runtime.h>
#include <hipblas.h>

namespace Dune
{
  namespace Cuda
  {
    //-------------- mv ---------------
    template <typename DT_>
    __global__ void device_mv(const DT_ * x, DT_ * y, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      const unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      const unsigned long row = idx;
      if (row >= rows)
        return;

      DT_ r(0);
      const unsigned long chunk(row / rows_per_chunk);
      const unsigned long local_row(row % rows_per_chunk);
      const unsigned long chunk_end( (chunk == chunks - 1) ? allocated_size : cs[chunk+1]);

      for (unsigned long pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol += rows_per_chunk)
          r += x[col[pcol]] * data[pcol];
      y[row] = r;
    }

    template <typename DT_>
    void mv(const DT_ * x, DT_ * y, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((rows)/(double)(block.x));
      device_mv<<<grid, block>>>(x, y, data, cs, col, rows, rows_per_chunk, chunks, allocated_size);
    }

    template void mv(const float*, float*, const float*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);
    template void mv(const double*, double*, const double*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);

    //-------------- umv ---------------
    template <typename DT_>
    __global__ void device_umv(const DT_ * x, DT_ * y, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      const unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      const unsigned long row = idx;
      if (row >= rows)
        return;

      DT_ r(y[row]);
      const unsigned long chunk(row / rows_per_chunk);
      const unsigned long local_row(row % rows_per_chunk);
      const unsigned long chunk_end( (chunk == chunks - 1) ? allocated_size : cs[chunk+1]);

      for (unsigned long pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol += rows_per_chunk)
          r += x[col[pcol]] * data[pcol];
      y[row] = r;
    }

    template <typename DT_>
    void umv(const DT_ * x, DT_ * y, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((rows)/(double)(block.x));
      device_umv<<<grid, block>>>(x, y, data, cs, col, rows, rows_per_chunk, chunks, allocated_size);
    }

    template void umv(const float*, float*, const float*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);
    template void umv(const double*, double*, const double*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);

    //-------------- mmv ---------------
    template <typename DT_>
    __global__ void device_mmv(const DT_ * x, DT_ * y, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      const unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      const unsigned long row = idx;
      if (row >= rows)
        return;

      DT_ r(y[row]);
      const unsigned long chunk(row / rows_per_chunk);
      const unsigned long local_row(row % rows_per_chunk);
      const unsigned long chunk_end( (chunk == chunks - 1) ? allocated_size : cs[chunk+1]);

      for (unsigned long pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol += rows_per_chunk)
          r -= x[col[pcol]] * data[pcol];
      y[row] = r;
    }

    template <typename DT_>
    void mmv(const DT_ * x, DT_ * y, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((rows)/(double)(block.x));
      device_mmv<<<grid, block>>>(x, y, data, cs, col, rows, rows_per_chunk, chunks, allocated_size);
    }

    template void mmv(const float*, float*, const float*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);
    template void mmv(const double*, double*, const double*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);

    //-------------- usmv ---------------
    template <typename DT_>
    __global__ void device_usmv(DT_ alpha, const DT_ * x, DT_ * y, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      const unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      const unsigned long row = idx;
      if (row >= rows)
        return;

      DT_ r(y[row]);
      const unsigned long chunk(row / rows_per_chunk);
      const unsigned long local_row(row % rows_per_chunk);
      const unsigned long chunk_end( (chunk == chunks - 1) ? allocated_size : cs[chunk+1]);

      for (unsigned long pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol += rows_per_chunk)
          r += x[col[pcol]] * data[pcol];
      y[row] = r * alpha;
    }

    template <typename DT_>
    void usmv(DT_ alpha, const DT_ * x, DT_ * y, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((rows)/(double)(block.x));
      device_usmv<<<grid, block>>>(alpha, x, y, data, cs, col, rows, rows_per_chunk, chunks, allocated_size);
    }

    template void usmv(float, const float*, float*, const float*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);
    template void usmv(double, const double*, double*, const double*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);
  }
}
