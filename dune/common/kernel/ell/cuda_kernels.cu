// -*- tab-width: 4; indent-tabs-mode: nil; c-basic-offset: 2 -*-
// vi: set et ts=4 sw=2 sts=2:


#include <hip/hip_runtime.h>
#include <hipblas.h>

namespace Dune
{
  namespace Cuda
  {
    //-------------- mv ---------------
    template <typename DT_>
    __global__ void device_mv(DT_ * y, const DT_ * x, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      const unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      const unsigned long row = idx;
      if (row >= rows)
        return;

      DT_ r(0);
      const unsigned long chunk(row / rows_per_chunk);
      const unsigned long local_row(row % rows_per_chunk);
      const unsigned long chunk_end( (chunk == chunks - 1) ? allocated_size : cs[chunk+1]);

      for (unsigned long pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol += rows_per_chunk)
          r += x[col[pcol]] * data[pcol];
      y[row] = r;
    }

    template <typename DT_>
    void mv(DT_ * y, const DT_ * x, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((rows)/(double)(block.x));
      device_mv<<<grid, block>>>(y, x, data, cs, col, rows, rows_per_chunk, chunks, allocated_size);
    }

    template void mv(float*, const float*, const float*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);
    template void mv(double*, const double*, const double*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);

    //-------------- umv ---------------
    template <typename DT_>
    __global__ void device_umv(DT_ * y, const DT_ * x, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      const unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      const unsigned long row = idx;
      if (row >= rows)
        return;

      DT_ r(y[row]);
      const unsigned long chunk(row / rows_per_chunk);
      const unsigned long local_row(row % rows_per_chunk);
      const unsigned long chunk_end( (chunk == chunks - 1) ? allocated_size : cs[chunk+1]);

      for (unsigned long pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol += rows_per_chunk)
          r += x[col[pcol]] * data[pcol];
      y[row] = r;
    }

    template <typename DT_>
    void umv(DT_ * y, const DT_ * x, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((rows)/(double)(block.x));
      device_umv<<<grid, block>>>(y, x, data, cs, col, rows, rows_per_chunk, chunks, allocated_size);
    }

    template void umv(float*, const float*, const float*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);
    template void umv(double*, const double*, const double*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);

    //-------------- mmv ---------------
    template <typename DT_>
    __global__ void device_mmv(DT_ * y, const DT_ * x, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      const unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      const unsigned long row = idx;
      if (row >= rows)
        return;

      DT_ r(y[row]);
      const unsigned long chunk(row / rows_per_chunk);
      const unsigned long local_row(row % rows_per_chunk);
      const unsigned long chunk_end( (chunk == chunks - 1) ? allocated_size : cs[chunk+1]);

      for (unsigned long pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol += rows_per_chunk)
          r -= x[col[pcol]] * data[pcol];
      y[row] = r;
    }

    template <typename DT_>
    void mmv(DT_ * y, const DT_ * x, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((rows)/(double)(block.x));
      device_mmv<<<grid, block>>>(y, x, data, cs, col, rows, rows_per_chunk, chunks, allocated_size);
    }

    template void mmv(float*, const float*, const float*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);
    template void mmv(double*, const double*, const double*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);

    //-------------- usmv ---------------
    template <typename DT_>
    __global__ void device_usmv(DT_ alpha, DT_ * y, const DT_ * x, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      const unsigned long idx = threadIdx.x + blockDim.x * blockIdx.x;
      const unsigned long row = idx;
      if (row >= rows)
        return;

      DT_ r(y[row]);
      const unsigned long chunk(row / rows_per_chunk);
      const unsigned long local_row(row % rows_per_chunk);
      const unsigned long chunk_end( (chunk == chunks - 1) ? allocated_size : cs[chunk+1]);

      for (unsigned long pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol += rows_per_chunk)
          r += x[col[pcol]] * data[pcol];
      y[row] = r * alpha;
    }

    template <typename DT_>
    void usmv(DT_ alpha, DT_ * y, const DT_ * x, const DT_ * data, const unsigned long * cs, const unsigned long * col,
        const unsigned long rows, const unsigned long rows_per_chunk,
        const unsigned long chunks, const unsigned long allocated_size)
    {
      unsigned long blocksize(128);
      dim3 grid;
      dim3 block;
      block.x = blocksize;
      grid.x = (unsigned)ceil((rows)/(double)(block.x));
      device_usmv<<<grid, block>>>(alpha, y, x, data, cs, col, rows, rows_per_chunk, chunks, allocated_size);
    }

    template void usmv(float, float*, const float*, const float*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);
    template void usmv(double, double*, const double*, const double*, const unsigned long*, const unsigned long*,
      const unsigned long, const unsigned long,
      const unsigned long, const unsigned long);
  }
}
